#include "hip/hip_runtime.h"
#include <Timer.hpp>
#include <iostream>
#include <iomanip>

using LOFAR::NSTimer;
using std::cout;
using std::cerr;
using std::endl;
using std::fixed;
using std::setprecision;

const int HISTOGRAM_SIZE = 256;
const unsigned int B_WIDTH = 32;
const unsigned int B_HEIGHT = 8;
//const int WARP_SIZE = 32;
//const int WARPS=8;

__global__ void histogram1DKernel(const int width, const int height, const unsigned char *inputImage, unsigned char *grayImage, unsigned int *histogram) {

    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int globalIdx=(blockDim.x * gridDim.x * i)+j;

    __shared__ unsigned int localHistogram[HISTOGRAM_SIZE];
    unsigned int inBlockIdx = threadIdx.x + (blockDim.x * threadIdx.y);
    localHistogram[inBlockIdx] = 0;
    __syncthreads();



    //unsigned int globalIdx = j + (width * i);
    //unsigned int warpid = inBlockIdx / WARP_SIZE;
    //unsigned int inWarpId = inBlockIdx % WARP_SIZE;

    if(globalIdx<width*height) {

        float grayPix = 0.0f;
        //if(blockIdx.x >= 10) {
        float r = static_cast< float >(inputImage[globalIdx]);
        float g = static_cast< float >(inputImage[(width * height) + globalIdx]);
        float b = static_cast< float >(inputImage[(2 * width * height) + globalIdx]);

        grayPix = ((0.3f * r) + (0.59f * g) + (0.11f * b)) + 0.5f;
        //}
        grayImage[(i * width) + j] = static_cast< unsigned char >(grayPix);

        atomicAdd((unsigned int *)&localHistogram[static_cast< unsigned int >(grayPix)], 1);
    }
     __syncthreads();
    atomicAdd((unsigned int *)&histogram[inBlockIdx], localHistogram[inBlockIdx]);

}



int histogram1D(const int width, const int height, const unsigned char *inputImage, unsigned char *grayImage, unsigned int *histogram) {
    hipError_t devRetVal = hipSuccess;
    unsigned char *devInputImage = 0;
    unsigned char *devGrayImage = 0;
    unsigned int *devHistogram = 0;

    int pixel_numbers;

    NSTimer globalTimer("GlobalTimer", false, false);
    NSTimer kernelTimer("KernelTimer", false, false);
    NSTimer memoryTimer("MemoryTimer", false, false);


    pixel_numbers = width * height;

    // Start of the computation
    globalTimer.start();
    // Convert the input image to grayscale and make it darker
    //*outputImage = new unsigned char[pixel_numbers];

    //cout << "FUNC2\n";
    // Allocate CUDA memory
    if ( (devRetVal = hipMalloc(reinterpret_cast< void ** >(&devInputImage), pixel_numbers * 3 * sizeof(unsigned char))) != hipSuccess ) {
        cerr << "Impossible to allocate device memory for inputImage." << endl;
        return 1;
    }
    if ( (devRetVal = hipMalloc(reinterpret_cast< void ** >(&devGrayImage), pixel_numbers * sizeof(unsigned char))) != hipSuccess ) {
        cerr << "Impossible to allocate device memory for darkGrayImage." << endl;
        return 1;
    }

    if ( (devRetVal = hipMalloc(reinterpret_cast< void ** >(&devHistogram), HISTOGRAM_SIZE * sizeof(unsigned int))) != hipSuccess ) {
        cerr << "Impossible to allocate device memory for histogram." << endl;
        return 1;
    }




    // Copy input to device
    memoryTimer.start();
    if ( (devRetVal = hipMemcpy(devInputImage, (void *)(inputImage), pixel_numbers * 3 * sizeof(unsigned char), hipMemcpyHostToDevice)) != hipSuccess ) {
        cerr << "Impossible to copy inputImage to device." << endl;
        return 1;
    }

    if ( (devRetVal = hipMemcpy(devHistogram, (void *)(histogram), HISTOGRAM_SIZE * sizeof(unsigned int), hipMemcpyHostToDevice)) != hipSuccess ) {
        cerr << "Impossible to copy inputImage to device." << endl;
        return 1;
    }

    /*if ( (devRetVal = hipMemcpy(devDarkGrayImage, reinterpret_cast< void *>(*outputImage), pixel_numbers * sizeof(unsigned char), hipMemcpyHostToDevice)) != hipSuccess ) {
        cerr << "Impossible to copy outputImage to device." << endl;
        return 1;
    }*/
    memoryTimer.stop();

    //cout << "FUNC4\n";
    //int grid_width = width % B_WIDTH == 0 ? width / B_WIDTH : width / B_WIDTH + 1;
    //int grid_height = height % B_HEIGHT == 0 ? height / B_HEIGHT : height / B_HEIGHT + 1;

    //cout << "Image size (w,h): (" << width << ", " << height << ")\n";
    //cout << "Grid size (w,h): (" << grid_width << ", " << grid_height << ")\n";

    unsigned int grid_width = static_cast< unsigned int >(ceil(width / static_cast< float >(B_WIDTH)));
    unsigned int grid_height = static_cast< unsigned int >(ceil(height / static_cast< float >(B_HEIGHT)));
    // Execute the kernel
    dim3 gridSize(grid_width, grid_height);
    dim3 blockSize(B_WIDTH, B_HEIGHT);
    cout << "gridsize: "<<grid_height<<"x"<<grid_width<<" -> "<<(grid_width*grid_height*256)<<" threads"<<endl;

    kernelTimer.start();
    //cout << "FUNC5\n";
    histogram1DKernel <<< gridSize, blockSize >>>(width, height, devInputImage, devGrayImage, devHistogram);
    hipDeviceSynchronize();
    kernelTimer.stop();
    //cout << "FUNC6\n";
    // Check if the kernel returned an error
    if ( (devRetVal = hipGetLastError()) != hipSuccess ) {
        cerr << "Uh, the kernel had some kind of issue: " << hipGetErrorString(devRetVal) << endl;
        return 1;
    }
    //cout << "FUNC7\n";
    // Copy the output back to host
    memoryTimer.start();
    if ( (devRetVal = hipMemcpy(reinterpret_cast< void *>(grayImage), devGrayImage, pixel_numbers * sizeof(unsigned char), hipMemcpyDeviceToHost)) != hipSuccess ) {
        cerr << "Impossible to copy devC to host." << endl;
        return 1;
    }
    if ( (devRetVal = hipMemcpy(reinterpret_cast< void *>(histogram), devHistogram, HISTOGRAM_SIZE * sizeof(unsigned int), hipMemcpyDeviceToHost)) != hipSuccess ) {
        cerr << "Impossible to copy devC to host." << endl;
        return 1;
    }
    memoryTimer.stop();

    globalTimer.stop();
    //cout << "FUNC8\n";
    //darkGrayImage._data = outputImage;
    // Time GFLOP/s GB/s
    cout << fixed << setprecision(6) << kernelTimer.getElapsed() << setprecision(3) << " " << (static_cast< long long unsigned int >(width) * height * 6) / 1000000000.0 / kernelTimer.getElapsed() << " " << (static_cast< long long unsigned int >(width) * height * ((4 * sizeof(unsigned char)) + (1 * sizeof(unsigned int)))) / 1000000000.0 / kernelTimer.getElapsed() << endl;


    // Print the timers
    cout << "Total (s): \t" << globalTimer.getElapsed() << endl;
    cout << "Kernel (s): \t" << kernelTimer.getElapsed() << endl;
    cout << "Memory (s): \t" << memoryTimer.getElapsed() << endl;
    cout << endl;

    hipFree(devInputImage);
    hipFree(devGrayImage);
    return 0;
}
