#include "hip/hip_runtime.h"
#include <Timer.hpp>
#include <iostream>
#include <iomanip>
#include <CImg.h>
#include <string>
#include <cmath>

using LOFAR::NSTimer;
using std::cout;
using std::cerr;
using std::endl;
using std::fixed;
using std::setprecision;
using cimg_library::CImg;
using std::string;

/*const unsigned int DIM = 16000000;
const unsigned int nrThreads = 256;*/
const unsigned int B_WIDTH = 16;
const unsigned int B_HEIGHT = 16;
consf int PIXELS_THREAD

__global__ void darkGrayKernel(const int width, const int height, const unsigned char *inputImage, unsigned char *darkGrayImage) {
    /*unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;*/

    //M[i,j]
    /*unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;*/

    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int globalIdx = j + (blockDim.x * gridDim.x * i);
    int k;

    for(k = 0; k < PIXELS_THREAD; k++) {
        if(globalIdx >= width * height) return;
        //if(j >= width || i >= height) return;
        float grayPix = 0.0f;
        float r = static_cast< float >(inputImage[globalIdx]);
        float g = static_cast< float >(inputImage[(width * height) + globalIdx]);
        float b = static_cast< float >(inputImage[(2 * width * height) + globalIdx]);
        grayPix = ((0.3f * r) + (0.59f * g) + (0.11f * b));
        grayPix = (grayPix * 0.6f) + 0.5f;
        //}
        darkGrayImage[globalIdx] = static_cast< unsigned char >(grayPix);
        globalIdx += (gridDim.x * blockDim.x) * (gridDim.y * blockDim.y);
    }



    int darkGray(const int width, const int height, const unsigned char *inputImage, unsigned char *darkGrayImage) {
        //cout << "FUNC\n";
        hipError_t devRetVal = hipSuccess;
        unsigned char *devInputImage = 0;
        unsigned char *devDarkGrayImage = 0;
        int pixel_numbers;

        NSTimer globalTimer("GlobalTimer", false, false);
        NSTimer kernelTimer("KernelTimer", false, false);
        NSTimer memoryTimer("MemoryTimer", false, false);

        int i, j;
        /*for(i = 0; i < width * height; i++) {
            cout << inputImage;
        }*/

        //cout << "FUNC1\n";
        pixel_numbers = width * height;

        // Start of the computation
        globalTimer.start();
        // Convert the input image to grayscale and make it darker
        //*outputImage = new unsigned char[pixel_numbers];

        //cout << "FUNC2\n";
        // Allocate CUDA memory
        if ( (devRetVal = hipMalloc(reinterpret_cast< void ** >(&devInputImage), pixel_numbers * 3 * sizeof(unsigned char))) != hipSuccess ) {
            cerr << "Impossible to allocate device memory for inputImage." << endl;
            return 1;
        }
        if ( (devRetVal = hipMalloc(reinterpret_cast< void ** >(&devDarkGrayImage), pixel_numbers * sizeof(unsigned char))) != hipSuccess ) {
            cerr << "Impossible to allocate device memory for darkGrayImage." << endl;
            return 1;
        }
        //cout << "FUNC3\n";

        // Copy input to device
        memoryTimer.start();
        if ( (devRetVal = hipMemcpy(devInputImage, (void *)(inputImage), pixel_numbers * 3 * sizeof(unsigned char), hipMemcpyHostToDevice)) != hipSuccess ) {
            cerr << "Impossible to copy inputImage to device." << endl;
            return 1;
        }

        /*if ( (devRetVal = hipMemcpy(devDarkGrayImage, reinterpret_cast< void *>(*outputImage), pixel_numbers * sizeof(unsigned char), hipMemcpyHostToDevice)) != hipSuccess ) {
            cerr << "Impossible to copy outputImage to device." << endl;
            return 1;
        }*/
        memoryTimer.stop();

        //cout << "FUNC4\n";
        //int grid_width = width % B_WIDTH == 0 ? width / B_WIDTH : width / B_WIDTH + 1;
        //int grid_height = height % B_HEIGHT == 0 ? height / B_HEIGHT : height / B_HEIGHT + 1;

        //cout << "Image size (w,h): (" << width << ", " << height << ")\n";
        //cout << "Grid size (w,h): (" << grid_width << ", " << grid_height << ")\n";

        //unsigned int grid_width=static_cast< unsigned int >(ceil(width / static_cast< float >(B_WIDTH)));
        //unsigned int grid_height=static_cast< unsigned int >(ceil(height / static_cast< float >(B_HEIGHT)));
        // Execute the kernel
        unsigned int grid_size = static_cast< unsigned int >(ceil(sqrt(ceil(width * height / PIXELS_THREAD) / (float)THREAD_NUMBER)));

        dim3 gridSize(grid_size, grid_size);
        dim3 blockSize(B_WIDTH, B_HEIGHT);

        kernelTimer.start();
        //cout << "FUNC5\n";
        darkGrayKernel <<< gridSize, blockSize >>>(width, height, devInputImage, devDarkGrayImage);
        hipDeviceSynchronize();
        kernelTimer.stop();
        //cout << "FUNC6\n";
        // Check if the kernel returned an error
        if ( (devRetVal = hipGetLastError()) != hipSuccess ) {
            cerr << "Uh, the kernel had some kind of issue: " << hipGetErrorString(devRetVal) << endl;
            return 1;
        }
        //cout << "FUNC7\n";
        // Copy the output back to host
        memoryTimer.start();
        if ( (devRetVal = hipMemcpy(reinterpret_cast< void *>(darkGrayImage), devDarkGrayImage, pixel_numbers * sizeof(unsigned char), hipMemcpyDeviceToHost)) != hipSuccess ) {
            cerr << "Impossible to copy devC to host." << endl;
            return 1;
        }
        memoryTimer.stop();

        globalTimer.stop();
        //cout << "FUNC8\n";
        //darkGrayImage._data = outputImage;
        // Time GFLOP/s GB/s
        cout << fixed << setprecision(6) << kernelTimer.getElapsed() << setprecision(3) << " " << (static_cast< long long unsigned int >(width) * height * 7) / 1000000000.0 / kernelTimer.getElapsed() << " " << (static_cast< long long unsigned int >(width) * height * (4 * sizeof(unsigned char))) / 1000000000.0 / kernelTimer.getElapsed() << endl;


        // Print the timers
        cout << "Total (s): \t" << globalTimer.getElapsed() << endl;
        cout << "Kernel (s): \t" << kernelTimer.getElapsed() << endl;
        cout << "Memory (s): \t" << memoryTimer.getElapsed() << endl;
        cout << endl;

        hipFree(devInputImage);
        hipFree(devDarkGrayImage);
        return 0;
    }