#include "hip/hip_runtime.h"

#include <Timer.hpp>
#include <iostream>
#include <iomanip>

using LOFAR::NSTimer;
using std::cout;
using std::cerr;
using std::endl;
using std::fixed;
using std::setprecision;

const unsigned int B_WIDTH = 32;
const unsigned int B_HEIGHT = 16;

__constant__ float filter[] = {1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 2.0f, 2.0f, 2.0f, 1.0f, 1.0f, 2.0f, 3.0f, 2.0f, 1.0f, 1.0f, 2.0f, 2.0f, 2.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f};


__global__ void triangularSmoothDKernel(const int width, const int height, const int spectrum, unsigned char *inputImage, unsigned char *smoothImage) {

    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if(j >= width || i >= height) return;

    for ( int z = 0; z < spectrum; z++ ) {
        unsigned int filterItem = 0;
        float filterSum = 0.0f;
        float smoothPix = 0.0f;

        for (int fy = i - 2; fy < i + 3; fy++ ) {
            if ( fy < 0 ) {
                filterItem += 5;
                continue;
            } else if ( fy == height ) {
                break;
            }

            for ( int fx = j - 2; fx < j + 3; fx++ ) {
                if ( (fx < 0) || (fx >= width) ) {
                    filterItem++;
                    continue;
                }

                smoothPix += static_cast< float >(inputImage[(z * width * height) + (fy * width) + fx]) * filter[filterItem];
                filterSum += filter[filterItem];
                filterItem++;
            }
        }

        smoothPix /= filterSum;
        smoothImage[(z * width * height) + (i * width) + j] = static_cast< unsigned char >(smoothPix + 0.5f);
    }
}


int triangularSmooth(const int width, const int height, const int spectrum, unsigned char *inputImage, unsigned char *smoothImage) {
    hipError_t devRetVal = hipSuccess;
    unsigned char *devInputImage = 0;
    unsigned char *devSmoothImage = 0;

    int pixel_numbers;

    NSTimer globalTimer("GlobalTimer", false, false);
    NSTimer kernelTimer("KernelTimer", false, false);
    NSTimer memoryTimer("MemoryTimer", false, false);


    pixel_numbers = width * height;

    // Start of the computation
    globalTimer.start();
    // Convert the input image to grayscale and make it darker
    //*outputImage = new unsigned char[pixel_numbers];

    //cout << "FUNC2\n";
    // Allocate CUDA memory
    if ( (devRetVal = hipMalloc(reinterpret_cast< void ** >(&devInputImage), pixel_numbers * 3 * sizeof(unsigned char))) != hipSuccess ) {
        cerr << "Impossible to allocate device memory for inputImage." << endl;
        return 1;
    }
    if ( (devRetVal = hipMalloc(reinterpret_cast< void ** >(&devSmoothImage), pixel_numbers * 3 * sizeof(unsigned char))) != hipSuccess ) {
        cerr << "Impossible to allocate device memory for darkGrayImage." << endl;
        return 1;
    }



    // Copy input to device
    memoryTimer.start();
    if ( (devRetVal = hipMemcpy(devInputImage, (void *)(inputImage), pixel_numbers * 3 * sizeof(unsigned char), hipMemcpyHostToDevice)) != hipSuccess ) {
        cerr << "Impossible to copy inputImage to device." << endl;
        return 1;
    }

    memoryTimer.stop();

    unsigned int grid_width = static_cast< unsigned int >(ceil(width / static_cast< float >(B_WIDTH)));
    unsigned int grid_height = static_cast< unsigned int >(ceil(height / static_cast< float >(B_HEIGHT)));
    // Execute the kernel
    dim3 gridSize(grid_width, grid_height);
    dim3 blockSize(B_WIDTH, B_HEIGHT);

    kernelTimer.start();
    triangularSmoothDKernel <<< gridSize, blockSize >>>(width, height, spectrum, devInputImage, devSmoothImage);
    hipDeviceSynchronize();
    kernelTimer.stop();
    //cout << "FUNC6\n";
    // Check if the kernel returned an error
    if ( (devRetVal = hipGetLastError()) != hipSuccess ) {
        cerr << "Uh, the kernel had some kind of issue: " << hipGetErrorString(devRetVal) << endl;
        return 1;
    }
    //cout << "FUNC7\n";
    // Copy the output back to host
    memoryTimer.start();
    if ( (devRetVal = hipMemcpy(reinterpret_cast< void *>(smoothImage), devSmoothImage, pixel_numbers * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost)) != hipSuccess ) {
        cerr << "Impossible to copy devC to host." << endl;
        return 1;
    }
    memoryTimer.stop();

    globalTimer.stop();
    //cout << "FUNC8\n";
    //darkGrayImage._data = outputImage;
    // Time GFLOP/s GB/s
    cout << fixed << setprecision(6) << kernelTimer.getElapsed() << setprecision(3) << " " << (static_cast< long long unsigned int >(width) * height * 6) / 1000000000.0 / kernelTimer.getElapsed() << " " << (static_cast< long long unsigned int >(width) * height * ((4 * sizeof(unsigned char)) + (1 * sizeof(unsigned int)))) / 1000000000.0 / kernelTimer.getElapsed() << endl;


    // Print the timers
    cout << "Total (s): \t" << globalTimer.getElapsed() << endl;
    cout << "Kernel (s): \t" << kernelTimer.getElapsed() << endl;
    cout << "Memory (s): \t" << memoryTimer.getElapsed() << endl;
    cout << endl;

    hipFree(devInputImage);
    hipFree(devSmoothImage);
    return 0;
}
