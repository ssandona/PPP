#include "hip/hip_runtime.h"
#include <Timer.hpp>
#include <iostream>
#include <iomanip>
#include <CImg.h>
#include <string>
#include <cmath>

using LOFAR::NSTimer;
using std::cout;
using std::cerr;
using std::endl;
using std::fixed;
using std::setprecision;
using cimg_library::CImg;
using std::string;

/*const unsigned int DIM = 16000000;
const unsigned int nrThreads = 256;*/
const unsigned int B_WIDTH = 16;
const unsigned int B_HEIGHT = 16;

__global__ void darkGrayKernel(unsigned int width, unsigned int height, unsigned char *inputImage, unsigned char *outputImage) {
    /*unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;*/

    //M[i,j]
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    
    if(j >= width || i >= height) return;

    float grayPix = 0.0f;
    float r = static_cast< float >(inputImage[(i * width) + j]);
    float g = static_cast< float >(inputImage[(width * height) + (i * width) + j]);
    float b = static_cast< float >(inputImage[(2 * width * height) + (i * width) + j]);

    grayPix = ((0.3f * r) + (0.59f * g) + (0.11f * b));
    grayPix = (grayPix * 0.6f) + 0.5f;

    if(blockIdx.x==0 && blockIdx.y==0){
        outputImage[(i * width) + j] = static_cast< unsigned char >(grayPix);
    }
    else {
        outputImage[(i * width) + j])=inputImage[(i * width) + j]);
        outputImage[(width * height) + (i * width) + j]=inputImage[(width * height) + (i * width) + j]
        outputImage[(2 * width * height) + (i * width) + j]=inputImage[(2 * width * height) + (i * width) + j];
    }
}



int darkGray(const int width, const int height, unsigned char *inputImage, unsigned char **outputImage) {
    cout << "FUNC\n";
    hipError_t devRetVal = hipSuccess;
    unsigned char *devInputImage=0;
    unsigned char *devDarkGrayImage=0;
    int pixel_numbers;

    NSTimer globalTimer("GlobalTimer", false, false);
    NSTimer kernelTimer("KernelTimer", false, false);
    NSTimer memoryTimer("MemoryTimer", false, false);

    int i,j;
    for(i=0;i<width*height;i++){
        cout << inputImage;
    }

    cout << "FUNC1\n";
    pixel_numbers=width * height;

    // Start of the computation
    globalTimer.start();
    // Convert the input image to grayscale and make it darker
    //*outputImage = new unsigned char[pixel_numbers];

    cout << "FUNC2\n";
    // Allocate CUDA memory
    if ( (devRetVal = hipMalloc(reinterpret_cast< void ** >(&devInputImage), pixel_numbers * 3 * sizeof(unsigned char))) != hipSuccess ) {
        cerr << "Impossible to allocate device memory for inputImage." << endl;
        return 1;
    }
    if ( (devRetVal = hipMalloc(reinterpret_cast< void ** >(&devDarkGrayImage), pixel_numbers * 3 * sizeof(unsigned char))) != hipSuccess ) {
        cerr << "Impossible to allocate device memory for darkGrayImage." << endl;
        return 1;
    }
    cout << "FUNC3\n";

    // Copy input to device
    memoryTimer.start();
    if ( (devRetVal = hipMemcpy(devInputImage, reinterpret_cast< void *>(inputImage), pixel_numbers * sizeof(unsigned char), hipMemcpyHostToDevice)) != hipSuccess ) {
        cerr << "Impossible to copy inputImage to device." << endl;
        return 1;
    }

    /*if ( (devRetVal = hipMemcpy(devDarkGrayImage, reinterpret_cast< void *>(*outputImage), pixel_numbers * sizeof(unsigned char), hipMemcpyHostToDevice)) != hipSuccess ) {
        cerr << "Impossible to copy outputImage to device." << endl;
        return 1;
    }*/
    memoryTimer.stop();

    cout << "FUNC4\n";
    int grid_width = width % B_WIDTH == 0 ? width / B_WIDTH : width / B_WIDTH + 1;
    int grid_height = height % B_HEIGHT == 0 ? height / B_HEIGHT : height / B_HEIGHT + 1;

    cout << "Image size (w,h): (" << width << ", " << height << ")\n";
    cout << "Grid size (w,h): (" << grid_width << ", " << grid_height << ")\n";

    // Execute the kernel
    dim3 gridSize(grid_width, grid_height,1);
    dim3 blockSize(B_WIDTH,B_HEIGHT,1);
    kernelTimer.start();
    cout << "FUNC5\n";
    darkGrayKernel <<< gridSize, blockSize >>>(width, height, devInputImage, devDarkGrayImage);
    hipDeviceSynchronize();
    kernelTimer.stop();
    cout << "FUNC6\n";
    // Check if the kernel returned an error
    if ( (devRetVal = hipGetLastError()) != hipSuccess ) {
        cerr << "Uh, the kernel had some kind of issue: " << hipGetErrorString(devRetVal) << endl;
        return 1;
    }
    cout << "FUNC7\n";
    // Copy the output back to host
    memoryTimer.start();
    if ( (devRetVal = hipMemcpy(reinterpret_cast< void *>(*outputImage), devDarkGrayImage, pixel_numbers * sizeof(unsigned char), hipMemcpyDeviceToHost)) != hipSuccess ) {
        cerr << "Impossible to copy devC to host." << endl;
        return 1;
    }
    memoryTimer.stop();
    cout << "FUNC8\n";
    //darkGrayImage._data = outputImage;
    // Time GFLOP/s GB/s
    cout << fixed << setprecision(6) << kernelTimer.getElapsed() << setprecision(3) << " " << (static_cast< long long unsigned int >(width) * height * 7) / 1000000000.0 / kernelTimer.getElapsed() << " " << (static_cast< long long unsigned int >(width) * height * (4 * sizeof(unsigned char))) / 1000000000.0 / kernelTimer.getElapsed() << endl;
     

    // Print the timers
    cout << "Total (s): \t" << globalTimer.getElapsed() << endl;
    cout << "Kernel (s): \t" << kernelTimer.getElapsed() << endl;
    cout << "Memory (s): \t" << memoryTimer.getElapsed() << endl;
    cout << endl;
    globalTimer.stop();

    // Save output
    //darkGrayImage.save(("./" + string(argv[1]) + ".dark.seq.bmp").c_str());

    hipFree(devInputImage);
    hipFree(devDarkGrayImage);
    return 0;
}
/*
int main(int argc, char *argv[]) {
    hipError_t devRetVal = hipSuccess;
    CImg< unsigned char > inputImage;
    unsigned char *devInputImage;
    CImg< unsigned char > darkGrayImage;
    unsigned char *devDarkGrayImage;
    unsigned char *outputImage;
    int pixel_numbers;

    NSTimer globalTimer("GlobalTimer", false, false);
    NSTimer kernelTimer("KernelTimer", false, false);
    NSTimer memoryTimer("MemoryTimer", false, false);


    if ( argc != 2 ) {
        cerr << "Usage: " << argv[0] << " <filename>" << endl;
        return 1;
    }

    // Load the input image
    inputImage = CImg< unsigned char >(argv[1]);
    if ( inputImage.spectrum() != 3 ) {
        cerr << "The input must be a color image." << endl;
        return 1;
    }

    pixel_numbers = width() * height();

    // Start of the computation
    globalTimer.start();

    // Convert the input image to grayscale and make it darker
    outputImage = new unsigned char[pixel_numbers];

    // Allocate CUDA memory
    if ( (devRetVal = hipMalloc(reinterpret_cast< void ** >(&devInputImage), pixel_numbers * sizeof(unsigned char))) != hipSuccess ) {
        cerr << "Impossible to allocate device memory for inputImage." << endl;
        return 1;
    }
    if ( (devRetVal = hipMalloc(reinterpret_cast< void ** >(&devDarkGrayImage), pixel_numbers * sizeof(unsigned char))) != hipSuccess ) {
        cerr << "Impossible to allocate device memory for darkGrayImage." << endl;
        return 1;
    }

    // Copy input to device
    memoryTimer.start();
    if ( (devRetVal = hipMemcpy(devInputImage, reinterpret_cast< void * >(inputImage.data()), pixel_numbers * sizeof(unsigned char), hipMemcpyHostToDevice)) != hipSuccess ) {
        cerr << "Impossible to copy devA to device." << endl;
        return 1;
    }
    memoryTimer.stop();


    int grid_width = width() % B_WIDTH == 0 ? width() / B_WIDTH : width() / B_WIDTH + 1;
    int grid_height = width() % B_HEIGHT == 0 ? height() / B_HEIGHT : height() / B_HEIGHT + 1;

    // Execute the kernel
    dim3 gridSize(static_cast< unsigned int >(ceil(height() / static_cast< float >(B_HEIGHT))), static_cast< unsigned int >(ceil(width() / static_cast< float >(B_WIDTH))));
    dim3 blockSize(B_WIDTH * B_HEIGHT);

    kernelTimer.start();
    darkGray <<< gridSize, blockSize >>>(grid_height, grid_width, devInputImage, devDarkGrayImage);
    hipDeviceSynchronize();
    kernelTimer.stop();

    // Check if the kernel returned an error
    if ( (devRetVal = hipGetLastError()) != hipSuccess ) {
        cerr << "Uh, the kernel had some kind of issue: " << hipGetErrorString(devRetVal) << endl;
        return 1;
    }

    // Copy the output back to host
    memoryTimer.start();
    if ( (devRetVal = hipMemcpy(reinterpret_cast< void * >(outputImage), devDarkGrayImage, pixel_numbers * sizeof(unsigned char), hipMemcpyDeviceToHost)) != hipSuccess ) {
        cerr << "Impossible to copy devC to host." << endl;
        return 1;
    }
    memoryTimer.stop();
    darkGrayImage._data = outputImage;
    //CImg<float> darkGrayImage(matrix,width,height,1,1,true);

    // End of the computation
    globalTimer.stop();

    // Print the timers
    cout << fixed << setprecision(6);
    cout << endl;
    cout << "Total (s): \t" << globalTimer.getElapsed() << endl;
    cout << "Kernel (s): \t" << kernelTimer.getElapsed() << endl;
    cout << "Memory (s): \t" << memoryTimer.getElapsed() << endl;
    cout << endl;
    cout << setprecision(3);
    cout << "GFLOP/s: \t" << (pixel_numbers / kernelTimer.getElapsed()) / 1000000000.0 << endl;
    cout << "GB/s: \t\t" << ((12 * pixel_numbers) / kernelTimer.getElapsed()) / 1000000000.0 << endl;
    cout << endl;

    // Save output
    darkGrayImage.save(("./" + string(argv[1]) + ".dark.seq.bmp").c_str());

    hipFree(devInputImage);
    hipFree(devDarkGrayImage);
    free(outputImage);

    return 0;
}
*/